
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // we only want to access the first N elements 
  if(i < N){
      a[i] = initialValue;
  }
}

int main()
{

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  /*
   * Assume we have reason to want the number of threads
   * fixed at `256`: do not modify `threads_per_block`.
   */

  size_t threads_per_block = 256;

  /*
   * Solution is to create more blocks than needed
   */

  size_t number_of_blocks = (N + threads_per_block - 1) / threads_per_block;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  /*
   * Check to make sure all values in `a`, were initialized.
   */

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      hipFree(a);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}
