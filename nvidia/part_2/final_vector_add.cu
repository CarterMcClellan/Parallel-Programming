#include "hip/hip_runtime.h"
#include <stdio.h>

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */
__global__
void initWith(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{

  /* get device properties */
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);
   
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  
  // add gpu pre-fetching logic here
  hipMemPrefetchAsync(a, size, deviceId);        // Prefetch to GPU device.
  
  hipMemPrefetchAsync(b, size, deviceId);        // Prefetch to GPU device.
  
  hipMemPrefetchAsync(c, size, deviceId);        // Prefetch to GPU device.
  //
  
  size_t threadsPerBlock = 256;
  size_t numberOfBlocks = 32 * numberOfSMs;
  
  printf("numberOfBlocks: %lu\nthreadsPerBlock: %lu\n", numberOfBlocks, threadsPerBlock);

  initWith<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);
  
  hipDeviceSynchronize();
  hipError_t addVectorsErr;
  hipError_t asyncErr;

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  // prefetch for CPU memory
  hipMemPrefetchAsync(c, size, hipCpuDeviceId); 
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
