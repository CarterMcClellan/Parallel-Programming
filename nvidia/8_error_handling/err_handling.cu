
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * Add error handling to this source code to learn what errors
   * exist, and then correct them. Googling error messages may be
   * of service if actions for resolving them are not clear to you.
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipError_t err;
  err = hipMallocManaged(&a, size);
  if (err != hipSuccess)                           // `cudaSuccess` is provided by CUDA.
  {
      printf("Error1: %s\n", hipGetErrorString(err)); // `cudaGetErrorString` is provided by CUDA.
  }

  init(a, N);

  size_t threads_per_block = 1024;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();
  hipError_t err2;
  err2 = hipGetLastError(); // `cudaGetLastError` will return the error from above.
  if (err2 != hipSuccess){
      printf("Error2: %s\n", hipGetErrorString(err2));
  }


  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
