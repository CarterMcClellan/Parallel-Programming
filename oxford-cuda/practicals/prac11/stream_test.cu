/*

This is based on an example developed by Mark Harris for his NVIDIA blog:

http://devblogs.nvidia.com/parallelforall/gpu-pro-tip-cuda-7-streams-simplify-concurrency/

-- I have added some timing to it

*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>

const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    // initialise CUDA timing, and start timer

    float milli;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float  *h_data, *d_data;
    h_data = (float *) malloc(sizeof(float));
    hipMalloc(&d_data, sizeof(float));
    h_data[0] = 1.0f;

    // set up 8 streams

    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    // loop over 8 streams

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipMalloc(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // do a Memcpy and launch a dummy kernel on the default stream
        hipMemcpy(d_data,h_data,sizeof(float),hipMemcpyHostToDevice);
        kernel<<<1, 1>>>(d_data, 0);
    }

    // wait for completion of all kernels

    hipDeviceSynchronize();

    // stop timer and report execution time

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milli, start, stop);
    printf("execution time (ms): %f \n",milli);

    hipDeviceReset();

    return 0;
}
