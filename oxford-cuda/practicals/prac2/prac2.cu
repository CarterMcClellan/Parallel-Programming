#include "hip/hip_runtime.h"

////////////////////////////////////////////////////////////////////////
// GPU version of Monte Carlo algorithm using NVIDIA's CURAND library
////////////////////////////////////////////////////////////////////////

#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// CUDA global constants
////////////////////////////////////////////////////////////////////////

__constant__ int   N;
__constant__ float T, r, sigma, rho, alpha, dt, con1, con2;


////////////////////////////////////////////////////////////////////////
// kernel routine
////////////////////////////////////////////////////////////////////////


__global__ void pathcalc(float *d_z, float *d_v)
{
  float s1, s2, y1, y2, payoff;
  int   ind;

  // move array pointers to correct position

  // version 1
  ind = threadIdx.x + 2*N*blockIdx.x*blockDim.x;

  // version 2
  // ind = 2*N*threadIdx.x + 2*N*blockIdx.x*blockDim.x;


  // path calculation

  s1 = 1.0f;
  s2 = 1.0f;

  for (int n=0; n<N; n++) {
    y1   = d_z[ind];
    // version 1
    ind += blockDim.x;      // shift pointer to next element
    // version 2
    // ind += 1; 

    y2   = rho*y1 + alpha*d_z[ind];
    // version 1
    ind += blockDim.x;      // shift pointer to next element
    // version 2
    // ind += 1; 

    s1 = s1*(con1 + con2*y1);
    s2 = s2*(con1 + con2*y2);
  }

  // put payoff value into device array

  payoff = 0.0f;
  if ( fabs(s1-1.0f)<0.1f && fabs(s2-1.0f)<0.1f ) payoff = exp(-r*T);

  d_v[threadIdx.x + blockIdx.x*blockDim.x] = payoff;
}


////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){
    
  int     NPATH=960000, h_N=100;
  float   h_T, h_r, h_sigma, h_rho, h_alpha, h_dt, h_con1, h_con2;
  float  *h_v, *d_v, *d_z;
  double  sum1, sum2;

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory on host and device

  h_v = (float *)malloc(sizeof(float)*NPATH);

  checkCudaErrors( hipMalloc((void **)&d_v, sizeof(float)*NPATH) );
  checkCudaErrors( hipMalloc((void **)&d_z, sizeof(float)*2*h_N*NPATH) );

  // define constants and transfer to GPU

  h_T     = 1.0f;
  h_r     = 0.05f;
  h_sigma = 0.1f;
  h_rho   = 0.5f;
  h_alpha = sqrt(1.0f-h_rho*h_rho);
  h_dt    = 1.0f/h_N;
  h_con1  = 1.0f + h_r*h_dt;
  h_con2  = sqrt(h_dt)*h_sigma;

  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(N),    &h_N,    sizeof(h_N)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(T),    &h_T,    sizeof(h_T)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(r),    &h_r,    sizeof(h_r)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(sigma),&h_sigma,sizeof(h_sigma)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(rho),  &h_rho,  sizeof(h_rho)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(alpha),&h_alpha,sizeof(h_alpha)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(dt),   &h_dt,   sizeof(h_dt)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con1), &h_con1, sizeof(h_con1)) );
  checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(con2), &h_con2, sizeof(h_con2)) );

  // random number generation

  hipEventRecord(start);

  hiprandGenerator_t gen;
  checkCudaErrors( hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT) );
  checkCudaErrors( hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL) );
  checkCudaErrors( hiprandGenerateNormal(gen, d_z, 2*h_N*NPATH, 0.0f, 1.0f) );
 
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("CURAND normal RNG  execution time (ms): %f,  samples/sec: %e \n",
          milli, 2.0*h_N*NPATH/(0.001*milli));

  // execute kernel and time it

  hipEventRecord(start);

  pathcalc<<<NPATH/64, 64>>>(d_z, d_v);
  getLastCudaError("pathcalc execution failed\n");

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);

  printf("Monte Carlo kernel execution time (ms): %f \n",milli);

  // copy back results

  checkCudaErrors( hipMemcpy(h_v, d_v, sizeof(float)*NPATH,
                   hipMemcpyDeviceToHost) );

  // compute average

  sum1 = 0.0;
  sum2 = 0.0;
  for (int i=0; i<NPATH; i++) {
    sum1 += h_v[i];
    sum2 += h_v[i]*h_v[i];
  }

  printf("\nAverage value and standard deviation of error  = %13.8f %13.8f\n\n",
	 sum1/NPATH, sqrt((sum2/NPATH - (sum1/NPATH)*(sum1/NPATH))/NPATH) );

  // Tidy up library

  checkCudaErrors( hiprandDestroyGenerator(gen) );

  // Release memory and exit cleanly

  free(h_v);
  checkCudaErrors( hipFree(d_v) );
  checkCudaErrors( hipFree(d_z) );

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

}
