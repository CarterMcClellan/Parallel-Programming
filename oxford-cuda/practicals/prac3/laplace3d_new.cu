#include "hip/hip_runtime.h"
//
// Program to solve Laplace equation on a regular 3D grid
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// define kernel block size
////////////////////////////////////////////////////////////////////////

#define BLOCK_X 16
#define BLOCK_Y 4
#define BLOCK_Z 4

////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////

#include <laplace3d_kernel_new.h>

////////////////////////////////////////////////////////////////////////
// declare Gold routine
////////////////////////////////////////////////////////////////////////

void Gold_laplace3d(int NX, int NY, int NZ, float* h_u1, float* h_u2);

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

  // 'h_' prefix - CPU (host) memory space

  int    NX=256, NY=256, NZ=256, REPEAT=10,
         bx, by, bz, i, j, k, ind;
  float  *h_u1, *h_u2, *h_u3, *h_foo, err;

  // 'd_' prefix - GPU (device) memory space

  float  *d_u1, *d_u2, *d_foo;

  printf("\nGrid dimensions: %d x %d x %d\n", NX, NY, NZ);

  // initialise card

  findCudaDevice(argc, argv);

  // initialise CUDA timing

  float milli;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // allocate memory for arrays

  h_u1 = (float *)malloc(sizeof(float)*NX*NY*NZ);
  h_u2 = (float *)malloc(sizeof(float)*NX*NY*NZ);
  h_u3 = (float *)malloc(sizeof(float)*NX*NY*NZ);
  checkCudaErrors( hipMalloc((void **)&d_u1, sizeof(float)*NX*NY*NZ) );
  checkCudaErrors( hipMalloc((void **)&d_u2, sizeof(float)*NX*NY*NZ) );

  // initialise u1

  for (k=0; k<NZ; k++) {
    for (j=0; j<NY; j++) {
      for (i=0; i<NX; i++) {
        ind = i + j*NX + k*NX*NY;

        if (i==0 || i==NX-1 || j==0 || j==NY-1|| k==0 || k==NZ-1)
          h_u1[ind] = 1.0f;           // Dirichlet b.c.'s
        else
          h_u1[ind] = 0.0f;
      }
    }
  }

  // copy u1 to device

  hipEventRecord(start);
  checkCudaErrors( hipMemcpy(d_u1, h_u1, sizeof(float)*NX*NY*NZ,
                              hipMemcpyHostToDevice) );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("\nCopy u1 to device: %.1f (ms) \n", milli);

  // Set up the execution configuration

  bx = 1 + (NX-1)/BLOCK_X;
  by = 1 + (NY-1)/BLOCK_Y;
  bz = 1 + (NZ-1)/BLOCK_Z;

  dim3 dimGrid(bx,by,bz);
  dim3 dimBlock(BLOCK_X,BLOCK_Y,BLOCK_Z);

  // printf("\n dimGrid  = %d %d %d \n",dimGrid.x,dimGrid.y,dimGrid.z);
  // printf(" dimBlock = %d %d %d \n",dimBlock.x,dimBlock.y,dimBlock.z);

  // Execute GPU kernel

  hipEventRecord(start);

  for (i = 1; i <= REPEAT; ++i) {
    GPU_laplace3d<<<dimGrid, dimBlock>>>(NX, NY, NZ, d_u1, d_u2);
    getLastCudaError("GPU_laplace3d execution failed\n");

    d_foo = d_u1; d_u1 = d_u2; d_u2 = d_foo;   // swap d_u1 and d_u2
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("\n%dx GPU_laplace3d_naive: %.1f (ms) \n", REPEAT, milli);

  // Read back GPU results

  hipEventRecord(start);
  checkCudaErrors( hipMemcpy(h_u2, d_u1, sizeof(float)*NX*NY*NZ,
                              hipMemcpyDeviceToHost) );
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("\nCopy u2 to host: %.1f (ms) \n", milli);

  // print out corner of array

  /*
  for (k=0; k<3; k++) {
    for (j=0; j<8; j++) {
      for (i=0; i<8; i++) {
        ind = i + j*NX + k*NX*NY;
        printf(" %5.2f ", h_u2[ind]);
      }
      printf("\n");
    }
    printf("\n");
  }
  */

  // Gold treatment

  hipEventRecord(start);
  for (int i = 1; i <= REPEAT; ++i) {
    Gold_laplace3d(NX, NY, NZ, h_u1, h_u3);
    h_foo = h_u1; h_u1 = h_u3; h_u3 = h_foo;   // swap h_u1 and h_u3
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milli, start, stop);
  printf("\n%dx Gold_laplace3d: %.1f (ms) \n \n", REPEAT, milli);

  // print out corner of array

  /*
  for (k=0; k<3; k++) {
    for (j=0; j<8; j++) {
      for (i=0; i<8; i++) {
        ind = i + j*NX + k*NX*NY;
        printf(" %5.2f ", h_u1[ind]);
      }
      printf("\n");
    }
    printf("\n");
  }
  */

  // error check

  err = 0.0;

  for (k=0; k<NZ; k++) {
    for (j=0; j<NY; j++) {
      for (i=0; i<NX; i++) {
        ind = i + j*NX + k*NX*NY;
        err += (h_u1[ind]-h_u2[ind])*(h_u1[ind]-h_u2[ind]);
      }
    }
  }

  printf("rms error = %f \n",sqrt(err/ (float)(NX*NY*NZ)));

 // Release GPU and CPU memory

  checkCudaErrors( hipFree(d_u1) );
  checkCudaErrors( hipFree(d_u2) );
  free(h_u1);
  free(h_u2);
  free(h_u3);

  hipDeviceReset();
}
