//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//
// template kernel routine
// 

template  <class T>
__global__ void my_first_kernel(T *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = threadIdx.x;
}


//
// CUDA routine to be called by main code
//

extern
int prac6(int nblocks, int nthreads)
{
  float *h_x, *d_x;
  int   *h_i, *d_i;
  int   nsize, n; 

  // allocate memory for arrays

  nsize = nblocks*nthreads ;

  h_x = (float *)malloc(nsize*sizeof(float));
  hipMalloc((void **)&d_x, nsize*sizeof(float));

  h_i = (int *)malloc(nsize*sizeof(int));
  hipMalloc((void **)&d_i, nsize*sizeof(int));

  // execute kernel for float

  my_first_kernel<<<nblocks,nthreads>>>(d_x);
  hipMemcpy(h_x,d_x,nsize*sizeof(float),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // execute kernel for ints

  my_first_kernel<<<nblocks,nthreads>>>(d_i);
  hipMemcpy(h_i,d_i,nsize*sizeof(int),hipMemcpyDeviceToHost);
  for (n=0; n<nsize; n++) printf(" n,  i  =  %d  %d \n",n,h_i[n]);

  // free memory 

  hipFree(d_x);
  free(h_x);
  hipFree(d_i);
  free(h_i);

  return 0;
}

 
