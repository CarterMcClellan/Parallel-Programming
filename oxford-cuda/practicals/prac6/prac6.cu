//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = threadIdx.x;
}


//
// CUDA routine to be called by main code
//

extern
int prac6(int nblocks, int nthreads)
{
  float *h_x, *d_x;
  int   nsize, n; 

  // allocate memory for arrays

  nsize = nblocks*nthreads ;

  h_x = (float *)malloc(nsize*sizeof(float));
  hipMalloc((void **)&d_x, nsize*sizeof(float));

  // execute kernel

  my_first_kernel<<<nblocks,nthreads>>>(d_x);

  // copy back results and print them out

  hipMemcpy(h_x,d_x,nsize*sizeof(float),hipMemcpyDeviceToHost);

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  hipFree(d_x);
  free(h_x);

  return 0;
}

 
