//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x;
}


//
// main code
//

int main(int argc, char **argv)
{
  float *h_x, *d_x;
  int   nblocks, nthreads, nsize, n; 

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  hipMalloc((void **)&d_x, nsize*sizeof(float));

  // execute kernel

  my_first_kernel<<<nblocks,nthreads>>>(d_x);

  // copy back results and print them out

  hipMemcpy(h_x,d_x,nsize*sizeof(float),hipMemcpyDeviceToHost);

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  hipFree(d_x);
  free(h_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
