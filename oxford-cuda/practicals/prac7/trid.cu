//
// Program to perform Backward Euler time-marching on a 1D grid
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////

#include <trid_kernel.h>

////////////////////////////////////////////////////////////////////////
// declare Gold routine
////////////////////////////////////////////////////////////////////////

void gold_trid(int, int, float*, float*);

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

  int    NX = 128, niter = 2;

  float *h_u, *h_v, *h_c, *d_u;

  // initialise card

  findCudaDevice(argc, argv);

  // allocate memory on host and device

  h_u = (float *)malloc(sizeof(float)*NX);
  h_v = (float *)malloc(sizeof(float)*NX);
  h_c = (float *)malloc(sizeof(float)*NX);

  checkCudaErrors( hipMalloc((void **)&d_u, sizeof(float)*NX) );

  // GPU execution

  for (int i=0; i<NX; i++) h_u[i] = 1.0f;

  checkCudaErrors( hipMemcpy(d_u, h_u, sizeof(float)*NX,
                              hipMemcpyHostToDevice) );

  GPU_trid<<<1, NX>>>(NX, niter, d_u);

  checkCudaErrors( hipMemcpy(h_u, d_u, sizeof(float)*NX,
                              hipMemcpyDeviceToHost) );


  // CPU execution

  for (int i=0; i<NX; i++) h_v[i] = 1.0f;

  gold_trid(NX, niter, h_v, h_c);


  // print out array

  for (int i=0; i<NX; i++) {
    printf(" %d  %f  %f  %f \n",i,h_u[i],h_v[i], h_u[i]-h_v[i]);
  }

 // Release GPU and CPU memory

  checkCudaErrors( hipFree(d_u) );

  free(h_u);
  free(h_v);
  free(h_c);

}
