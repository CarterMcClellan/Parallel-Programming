//
// Pattern-matching program
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////

// #include <match_kernel.cu>

////////////////////////////////////////////////////////////////////////
// declare Gold routine
////////////////////////////////////////////////////////////////////////

void gold_match(unsigned int *, unsigned int *, int *, int, int);

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

  char *ctext, *cwords[] = {"cuti", "gold", "text", "word"};
  unsigned int  *text,  *words;

  int   length, len, nwords=4, matches[4]={0, 0, 0, 0};

  // read in text for processing

  FILE *fp;
  fp = fopen("match.cu","r");

  length = 0;
  while (getc(fp) != EOF) length++;

  ctext = (char *) malloc(length+4);

  rewind(fp);

  for (int l=0; l<length; l++) ctext[l] = getc(fp);
  for (int l=length; l<length+4; l++) ctext[l] = ' ';

  fclose(fp);

  // define number of words of text, and set pointers

  len  = length/4;
  text = (unsigned int *) ctext;

  // define words for matching

  words = (unsigned int *) malloc(nwords*sizeof(unsigned int));

  for (int w=0; w<nwords; w++) {
    words[w] = ((unsigned int) cwords[w][0])
             + ((unsigned int) cwords[w][1])*256
             + ((unsigned int) cwords[w][2])*256*256
             + ((unsigned int) cwords[w][3])*256*256*256;
  }

  // CPU execution

  gold_match(text, words, matches, nwords, len);

  printf(" CPU matches = %d %d %d %d \n",
         matches[0],matches[1],matches[2],matches[3]);

  // GPU execution




  // Release GPU and CPU memory


  free(ctext);
}
