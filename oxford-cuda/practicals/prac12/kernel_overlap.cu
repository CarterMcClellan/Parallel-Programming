#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"

__global__ void do_work(double *data, int N, int idx) {
	int i = blockIdx.x * blockDim.x + blockDim.x*idx + threadIdx.x;
	if (i < N) {
		for (int j = 0; j < 200; j++) {
			data[i] = cos(data[i]);
			data[i] = sqrt(fabs(data[i]));
		}
	}
}

int main()
{
	int nblocks = 30;
	int blocksize = 1024;
	double *data;
	checkCudaErrors(hipMalloc( (void**)&data, nblocks*blocksize*sizeof(double) ));



	float time;
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));
	dim3 dimBlock( blocksize, 1, 1 );
	dim3 dimGrid( 1, 1, 1 );
	for (int i = 0; i < nblocks; i++)
		do_work<<<dimGrid,dimBlock>>>(data, nblocks*blocksize, i);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&time, start, stop));
	printf("Serialised time:  %g ms\n", time);

	hipStream_t streams[nblocks];
	for (int i = 0; i < nblocks; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));

	checkCudaErrors(hipEventRecord(start, 0));
	checkCudaErrors(hipEventSynchronize(start));
	for (int i = 0; i < nblocks; i++)
		do_work<<<dimGrid,dimBlock,0,streams[i]>>>(data, nblocks*blocksize, i);
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&time, start, stop));
	printf("Multi-stream parallel time:  %g ms\n", time);

	for (int i = 0; i < nblocks; i++)
		checkCudaErrors(hipStreamDestroy(streams[i]));

	checkCudaErrors(hipFree( data ));
	return EXIT_SUCCESS;
}

