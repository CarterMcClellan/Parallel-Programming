#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"

__global__ void do_work(double *data, int N, int idx) {
	int i = blockIdx.x * blockDim.x + blockDim.x*idx + threadIdx.x;
	if (i < N) {
		for (int j = 0; j < 20; j++) {
			data[i] = cos(data[i]);
			data[i] = sqrt(fabs(data[i]));
		}
	}
}

int main()
{
	//Allocate 1 GB of data
	int total_data = 1<<27;
	double *d_data, *h_data;
	checkCudaErrors(hipHostMalloc((void**)&h_data, total_data*sizeof(double)));
	checkCudaErrors(hipMalloc( (void**)&d_data, total_data*sizeof(double) ));

	//Initialise host data
	srand(0);
	for (int i = 0; i < total_data; i++)
		h_data[i] = (double)rand()/(double)RAND_MAX;

	int batches=8;
	hipStream_t streams[batches];
	for (int i = 0; i < batches; i++)
		checkCudaErrors(hipStreamCreate(&streams[i]));

	//Start timing	
	float time;
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	checkCudaErrors(hipEventRecord(start, 0));

	//Figure out how many blocks are needed
	int blocksize = 256;
	int data_fraction = (total_data-1)/batches+1;
	int nblocks = (data_fraction-1)/blocksize+1;

	for (int i = 0; i < batches; i++) {
		//Make sure we have the right size for each chunk
		int upload_size = data_fraction;
		if (i==batches-1) upload_size = total_data - data_fraction*i;

		//Copy data to device
		checkCudaErrors(hipMemcpyAsync(&d_data[i*data_fraction],&h_data[i*data_fraction],upload_size*sizeof(double),hipMemcpyHostToDevice,streams[i]));

		//Launch kernel to process data
		do_work<<<nblocks,blocksize,0,streams[i]>>>(d_data, total_data, i*nblocks);

		//Copy data back from device
		checkCudaErrors(hipMemcpyAsync(&h_data[i*data_fraction],&d_data[i*data_fraction],upload_size*sizeof(double),hipMemcpyDeviceToHost,streams[i]));
	}

	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&time, start, stop));
	printf("Total processing time:  %g ms\n", time);

	for (int i = 0; i < batches; i++)
		checkCudaErrors(hipStreamDestroy(streams[i]));
	checkCudaErrors(hipFree( d_data ));
	checkCudaErrors(hipHostFree(h_data));
	return EXIT_SUCCESS;
}

